#include "hip/hip_runtime.h"
#include <ctime> 
#include <cmath>
#include <cstdio>
#include <fstream>
#include <iostream>

#include "test_scene.h"
#include "mesh/obj_loader.h"
#include "material/material.h"

#define RESOLUTION 1
#define SAMPLES 100

#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)


void check_cuda(hipError_t result, 
                char const *const func, 
                const char *const file, 
                int const line) {
    if(result) {
        std::cerr << "CUDA error = "<< static_cast<unsigned int>(result) << " at " <<
        file << ":" << line << " '" << func << "' \n";
        hipDeviceReset();
        exit(99);
    }
}


__device__ vec3 shade(const Ray& r, 
                      Hitable **world, 
                      int depth,
                      hiprandState *state) {
    HitRecord rec;
    if((*world)->hit(r, 0.001, MAXFLOAT, rec)) {
        Ray scattered;
        vec3 attenuation;
        vec3 emitted = rec.mat_ptr->emitted(rec.u, rec.v, rec.p);
        if(depth < 15 && rec.mat_ptr->scatter(r, rec, attenuation, scattered, state)){
            return emitted + attenuation * shade(scattered, world, depth + 1, state);
        }else{
            return emitted;
        }
    }else{
        return vec3(0, 0, 0);
    }
}


__device__ vec3 shade_nolight(const Ray& r, 
                              Hitable **world, 
                              int depth,
                              hiprandState *state) {
    HitRecord rec;
    if((*world)->hit(r, 0.001, MAXFLOAT, rec)) {
        Ray scattered;
        vec3 attenuation;
        if(depth < 15 && rec.mat_ptr->scatter(r, rec, attenuation, scattered, state)) {
            return attenuation * shade_nolight(scattered, world, depth + 1, state);
        }
        else {
            return vec3(0, 0, 0);
        }
    }else{
        return vec3(1.0, 1.0, 1.0);
    }
}


__global__ void build_scene(Hitable** obj_list, 
                            Hitable** world,
                            Camera** camera,
                            hiprandState *state,
                            int nx,
                            int ny,
                            int cnt){
    if(threadIdx.x == 0 && blockIdx.x == 0) {

        vec3 lookfrom(0, 0, 20);
        vec3 lookat(0, 0, 0);
        float dist_to_focus = 10.0;
        float aperture = 0.0;
        float vfov = 60.0;

        *camera = new MotionCamera(lookfrom, 
                                lookat, 
                                vec3(0, 1, 0), 
                                vfov, 
                                float(nx) / float(ny), 
                                aperture, 
                                dist_to_focus,
                                0.0,
                                1.0);
    }
}

__global__ void build_mesh(Hitable** mesh,
                            Camera** camera,
                            Hitable** triangles,
                            vec3* points,
                            vec3* idxVertex,
                            int np, int nt,
                            hiprandState *state,
                            int nx, int ny, int cnt){
    if(threadIdx.x == 0 && blockIdx.x == 0) {

        draw_one_mesh(mesh, triangles, points, idxVertex, np, nt, state);
        // bunny_inside_cornell_box(mesh, triangles, points, idxVertex, np, nt, state);

        vec3 lookfrom(0, 0, 10);
        vec3 lookat(0, 0, 0);
        float dist_to_focus = 10.0;
        float aperture = 0.0;
        float vfov = 60.0;

        *camera = new MotionCamera(lookfrom, 
                                    lookat, 
                                    vec3(0, 1, 0), 
                                    vfov, 
                                    float(nx) / float(ny), 
                                    aperture, 
                                    dist_to_focus,
                                    0.0,
                                    1.0);
    }
}

__global__ void random_init(int nx, 
                            int ny, 
                            hiprandState *state) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if((x >= nx) || (y >= ny)) return;
    int pixel_index = y * nx + x;
    hiprand_init(0, pixel_index, 0, &state[pixel_index]);
}


__global__ void destroy(Hitable** obj_list, 
                        Hitable** world,
                        Camera** camera, 
                        int obj_cnt) {
    for(int i = 0; i < obj_cnt; i++) {
        delete *(obj_list + i);
    }
    delete *world;
    delete *camera;
}


__global__ void render(vec3* colorBuffer,
                       Hitable** world,
                       Camera** camera,
                       hiprandState* state,
                       int nx, 
                       int ny,
                       int samples) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if((x >= nx) || (y >= ny)) return;
    
    int pixel_index = y * nx + x;

    int ns = samples;
    vec3 col(0, 0, 0);
    for(int i = 0; i < ns; i++){
        float u = float(x + rand(&(state[pixel_index]))) / float(nx);
        float v = float(y + rand(&(state[pixel_index]))) / float(ny);
        Ray r = (*camera)->get_ray(u, v, state);
        col += shade(r, world, 0, &(state[pixel_index]));
        // col += shade_nolight(r, world, 0, &(state[pixel_index]));
    }
    col /= float(ns);
    col[0] = sqrt(col[0]);
    col[1] = sqrt(col[1]);
    col[2] = sqrt(col[2]);

    colorBuffer[pixel_index] = clip(col);
}


int main() {
    std::time_t tic = std::time(NULL);
    std::cout << "Start running at: " << std::asctime(std::localtime(&tic)) << std::endl;

    std::ofstream imgWrite("images/image.ppm");

    int nx = 1024 * RESOLUTION;
    int ny = 512  * RESOLUTION;
    int tx = 16;
    int ty = 16;
    
    int num_pixel = nx * ny;

    // color buffer
    vec3 *colorBuffer;
    checkCudaErrors(hipMallocManaged((void**)& colorBuffer, num_pixel*sizeof(vec3)));

    hiprandState* curand_state;
    checkCudaErrors(hipMallocManaged((void**)& curand_state, num_pixel*sizeof(hiprandState)));

    // build world
    int obj_cnt = 488;
    Hitable** obj_list;
    Hitable** world;
    Camera**  camera;
    checkCudaErrors(hipMallocManaged((void**)& obj_list, obj_cnt*sizeof(Hitable*)));
    checkCudaErrors(hipMallocManaged((void**)& world, sizeof(Hitable*)));
    checkCudaErrors(hipMallocManaged((void**)& camera, sizeof(Camera*)));

    dim3 blocks(nx / tx + 1, ny / ty + 1);
    dim3 threads(tx, ty);
    random_init <<<blocks, threads>>>(nx, ny, curand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // --------------------------- allocate the mesh ----------------------------------------
    vec3* points;
    vec3* idxVertex;

    // NOTE: must pre-allocate before initialize the elements
    checkCudaErrors(hipMallocManaged((void**)& points,    2600 * sizeof(vec3)));
    checkCudaErrors(hipMallocManaged((void**)& idxVertex, 5000 * sizeof(vec3)));

    int nPoints, nTriangles;
    parseObjByName("./shapes/small_bunny.obj", points, idxVertex, nPoints, nTriangles);

    std::cout << "# of points: " << nPoints << std::endl;
    std::cout << "# of triangles: " << nTriangles << std::endl;

    // scale
    for(int i = 0; i < nPoints; i++) { points[i] *= 30.0; }
    for(int i = 0; i < nPoints; i++) { std::cout << points[i] << std::endl; }

    Hitable** triangles;
    checkCudaErrors(hipMallocManaged((void**)& triangles, nTriangles * sizeof(Hitable*)));
    // --------------------------- ! allocate the mesh ---------------------------------------

    build_mesh <<<1, 1>>>(world, camera, triangles, points, 
                          idxVertex, nPoints, nTriangles, curand_state, nx, ny, obj_cnt);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    render <<<blocks, threads>>>(colorBuffer, world, camera, curand_state, nx, ny, SAMPLES);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    imgWrite << "P3\n" << nx << " " << ny << "\n255\n";
    for(int i = ny - 1; i >= 0; i--) {
        for(int j = 0; j < nx; j++) {
            size_t pixel_index = i * nx + j;
            int ir = int(255.99 * colorBuffer[pixel_index].r());
            int ig = int(255.99 * colorBuffer[pixel_index].g());
            int ib = int(255.99 * colorBuffer[pixel_index].b());
            imgWrite << ir << " " << ig << " " << ib << "\n";
        }
    }
    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    destroy<<<1, 1>>>(obj_list, world, camera, obj_cnt);

    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(world));
    checkCudaErrors(hipFree(obj_list));
    checkCudaErrors(hipFree(camera));
    checkCudaErrors(hipFree(curand_state));
    checkCudaErrors(hipFree(colorBuffer));

    hipDeviceReset();

    std::time_t toc = std::time(NULL);
    std::cout << "Finish running at: " << std::asctime(std::localtime(&toc)) << std::endl;
    std::cout << "Time consuming: " << toc - tic << "s" << std::endl;
}
